#include <nvgraph.h>
#include <stdio.h> 

void check(nvgraphStatus_t status) {
    if (status != NVGRAPH_STATUS_SUCCESS) {
        printf("ERROR : %d\n",status);
        exit(0);
    }
}
int main(int argc, char *argv[]) {
    /*Check Errors*/
    if(argc != 7){
        printf("Arguments should be in following order:\n");
        printf("<# Vertices> <# Edges> <Weights txt File> <Offset txt File> <Indices txt File> <Save Name>\n");
        return 1;
    }

    const size_t vertex_numsets = 1, edge_numsets = 1;
    size_t n, nnz;
    int i, *sssp_1_h_total;
    float *sssp_1_h;
    void** vertex_dim;

    /*Assign Variables*/
    n = atoi(argv[1]);
    nnz = atoi(argv[2]);

    /*Open Files*/
    FILE *weights, *offsets, *indices, *results;
    weights = fopen(argv[3], "r");
    offsets = fopen(argv[4], "r");
    indices = fopen(argv[5], "r");
    
    float weights_h[nnz];
    float destination_offsets[n+1];
    int destination_offsets_h[n+1];
    float source_indices[nnz];
    int source_indices_h[nnz];
    
    for (i = 0; i < nnz; i++) {
        /*Assign Weights*/
        fscanf(weights, "%e", &weights_h[i]);
        
        /*Assign Indices*/
        fscanf(indices, "%e", &source_indices[i]);
        source_indices_h[i] = source_indices[i];
    }

    for (i = 0; i < n; i++) {
        /*Assign Indices*/
        fscanf(offsets, "%e", &destination_offsets[i]);
        destination_offsets_h[i] = destination_offsets[i];
    }
    
    fclose(weights);
    fclose(indices);
    fclose(offsets);

    destination_offsets_h[n] = nnz;

    // nvgraph variables
    nvgraphStatus_t status; nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType* vertex_dimT;

    // Init host data
    sssp_1_h = (float*)malloc(n*sizeof(float));
    sssp_1_h_total = (int*)malloc(n*sizeof(int));
    for (i = 0; i < n; i++) {
        sssp_1_h_total[i] = 0;
    }
    vertex_dim = (void**)malloc(vertex_numsets*sizeof(void*));
    vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0]= (void*)sssp_1_h; vertex_dimT[0] = HIP_R_32F;

    check(nvgraphCreate(&handle));
    check(nvgraphCreateGraphDescr (handle, &graph));
    CSC_input->nvertices = n; 
    CSC_input->nedges = nnz;
    CSC_input->destination_offsets = destination_offsets_h;
    CSC_input->source_indices = source_indices_h;

    // Set graph connectivity and properties (tranfers)
    check(nvgraphSetGraphStructure(handle, graph, (void*)CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData (handle, graph, edge_numsets, &edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void*)weights_h, 0));

    // Solve
    int source_vert = 0;
    check(nvgraphSssp(handle, graph, 0, &source_vert, 0));

    // Get and print result
    check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));

    /*Find Diameter and Calculate SSSP Count*/
    int diameter = 0;
    for (i = 0; i < n; i++){
        if(sssp_1_h[i] < 3402823466){
            sssp_1_h_total[(int)sssp_1_h[i]]++;
            if(diameter < sssp_1_h[i]) diameter = sssp_1_h[i];
        }
    }

    printf("Diameter: %d\n", diameter);

    /*Write the Shortest Path to a file*/
    results = fopen(argv[6], "w+");
    for (i = 0; i <= diameter; i++){
        fprintf(results, "%d\n", sssp_1_h_total[i]);
    }
    fclose(results);

    //Clean 
    free(sssp_1_h); free(vertex_dim);
    free(vertex_dimT); free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));
    return 0;
}