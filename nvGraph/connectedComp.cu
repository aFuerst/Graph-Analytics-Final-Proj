#include <nvgraph.h>
#include <stdio.h> 

void check(nvgraphStatus_t status) {
    if (status != NVGRAPH_STATUS_SUCCESS) {
        printf("ERROR : %d\n",status);
        exit(0);
    }
}
int main(int argc, char *argv[]) {
    /*Check Errors*/
    if(argc != 7){
        printf("Arguments should be in following order:\n");
        printf("<# Vertices> <# Edges> <Weights txt File> <Offset txt File> <Indices txt File> <Save Name>\n");
        return 1;
    }

    const size_t vertex_numsets = 1, edge_numsets = 1;
    size_t n, nnz;
    int i;
    float *sssp_1_h;
    void** vertex_dim;

    /*Assign Variables*/
    n = atoi(argv[1]);
    nnz = atoi(argv[2]);

    /*Open Files*/
    FILE *weights, *offsets, *indices, *results;
    weights = fopen(argv[3], "r");
    offsets = fopen(argv[4], "r");
    indices = fopen(argv[5], "r");
    
    float weights_h[nnz];
    float destination_offsets[n+1];
    int destination_offsets_h[n+1];
    float source_indices[nnz];
    int source_indices_h[nnz];
    int are_the_vertex_not_checked[n];
    int connected_components[n];
    
    for (i = 0; i < nnz; i++) {
        /*Assign Weights*/
        fscanf(weights, "%e", &weights_h[i]);
        
        /*Assign Indices*/
        fscanf(indices, "%e", &source_indices[i]);
        source_indices_h[i] = source_indices[i];
    }

    for (i = 0; i < n; i++) {
        /*Assign Indices*/
        fscanf(offsets, "%e", &destination_offsets[i]);
        destination_offsets_h[i] = destination_offsets[i];
    }
    
    fclose(weights);
    fclose(indices);
    fclose(offsets);

    destination_offsets_h[n] = nnz;

    /*Initialize are_the_vertex_not_checked and connected_components*/
    for (i = 0; i < n; i++) {
        are_the_vertex_not_checked[i] = 0;
        connected_components[i] = 0;
    }

    // nvgraph variables
    nvgraphStatus_t status; nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType* vertex_dimT;

    // Init host data
    sssp_1_h = (float*)malloc(n*sizeof(float));
    vertex_dim = (void**)malloc(vertex_numsets*sizeof(void*));
    vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0]= (void*)sssp_1_h; vertex_dimT[0] = HIP_R_32F;

    check(nvgraphCreate(&handle));
    check(nvgraphCreateGraphDescr (handle, &graph));
    CSC_input->nvertices = n; 
    CSC_input->nedges = nnz;
    CSC_input->destination_offsets = destination_offsets_h;
    CSC_input->source_indices = source_indices_h;

    // Set graph connectivity and properties (tranfers)
    check(nvgraphSetGraphStructure(handle, graph, (void*)CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData (handle, graph, edge_numsets, &edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void*)weights_h, 0));

    // Solve
    int source_vert = 0;
    check(nvgraphSssp(handle, graph, 0, &source_vert, 0));

    // Get and print result
    check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));

    /*Fill are_the_vertex_not_checked and connected_components for the first time*/
    source_vert = -1;
    int location = 0;
    for (i = 0; i < n; i++) {
        if(sssp_1_h[i] > 3402823466) {
            if(source_vert == -1) source_vert = i;
            are_the_vertex_not_checked[i] = 1;
        }
        else connected_components[location]++;
    }
    
    while(1){
        /*Calculte SSSP*/
        check(nvgraphSssp(handle, graph, 0, &source_vert, 0));
        check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));

        location++;
        source_vert = -1;

        /*Calculate Connected Component Count*/
        for (i = 0; i < n; i++) {
            if(sssp_1_h[i] < 3402823466) {
                connected_components[location]++;
                are_the_vertex_not_checked[i] = 0;
            }
        }

        /*Find next source_vert*/
        for (i = 0; i < n; i++) {
            if(are_the_vertex_not_checked[i] == 1) {
                if(source_vert == -1) source_vert = i;
                break;
            }
        }

        if(source_vert == -1) break;
    }

    /*Write the Shortest Path to a file*/
    results = fopen(argv[6], "w+");
    for (int i = 0; i <= location; i++){
        fprintf(results, "%d\n", connected_components[i]);
    }
    fclose(results);

    //Clean 
    free(sssp_1_h); free(vertex_dim);
    free(vertex_dimT); free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));
    return 0;
}