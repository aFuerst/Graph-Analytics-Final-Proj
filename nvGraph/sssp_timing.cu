#include <nvgraph.h>
#include <stdio.h> 
#include <sys/time.h>
#include <time.h>

void check(nvgraphStatus_t status) {
    if (status != NVGRAPH_STATUS_SUCCESS) {
        printf("ERROR : %d\n",status);
        exit(0);
    }
}
int main(int argc, char *argv[]) {
    /*Check Errors*/
    if(argc != 7){
        printf("Arguments should be in following order:\n");
        printf("<# Vertices> <# Edges> <Weights txt File> <Offset txt File> <Indices txt File> <Save Name>\n");
        return 1;
    }

    const size_t vertex_numsets = 1, edge_numsets = 1;
    size_t n, nnz;
    int i;
    float *sssp_1_h;
    void** vertex_dim;

    /*Declare time based variables*/
    struct timeval tv1, tv2;
    struct timezone tz;
    long int total_time = 0, timing[100];
    
    /*Assign Variables*/
    n = atoi(argv[1]);
    nnz = atoi(argv[2]);

    /*Open Files*/
    FILE *weights, *offsets, *indices, *results;
    weights = fopen(argv[3], "r");
    offsets = fopen(argv[4], "r");
    indices = fopen(argv[5], "r");
    
    float weights_h[nnz];
    float destination_offsets[n+1];
    int destination_offsets_h[n+1];
    float source_indices[nnz];
    int source_indices_h[nnz];
    
    for (i = 0; i < nnz; i++) {
        /*Assign Weights*/
        fscanf(weights, "%e", &weights_h[i]);
        
        /*Assign Indices*/
        fscanf(indices, "%e", &source_indices[i]);
        source_indices_h[i] = source_indices[i];
    }

    for (i = 0; i < n; i++) {
        /*Assign Indices*/
        fscanf(offsets, "%e", &destination_offsets[i]);
        destination_offsets_h[i] = destination_offsets[i];
    }
    
    fclose(weights);
    fclose(indices);
    fclose(offsets);

    destination_offsets_h[n] = nnz;

    // nvgraph variables
    nvgraphStatus_t status; nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType* vertex_dimT;

    // Init host data
    sssp_1_h = (float*)malloc(n*sizeof(float));
    vertex_dim = (void**)malloc(vertex_numsets*sizeof(void*));
    vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0]= (void*)sssp_1_h; vertex_dimT[0] = HIP_R_32F;

    check(nvgraphCreate(&handle));
    check(nvgraphCreateGraphDescr (handle, &graph));
    CSC_input->nvertices = n; 
    CSC_input->nedges = nnz;
    CSC_input->destination_offsets = destination_offsets_h;
    CSC_input->source_indices = source_indices_h;

    // Set graph connectivity and properties (tranfers)
    check(nvgraphSetGraphStructure(handle, graph, (void*)CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData (handle, graph, edge_numsets, &edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void*)weights_h, 0));

    /*Measure timing for 100 different source_vert*/
    int source_vert_offset = n/100;
    int source_vert = 0;
    
    for (int i = 0; i < 100; i++){
        gettimeofday(&tv1,&tz); // Get starting time

        // Solve        
        check(nvgraphSssp(handle, graph, 0, &source_vert, 0));

        // Get and print result
        check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));

        gettimeofday(&tv2,&tz); // Get ending time

        source_vert = source_vert + source_vert_offset;
        /*Calculate time taken in microseconds*/
        timing[i] = (tv2.tv_sec-tv1.tv_sec)*1000000 + (tv2.tv_usec-tv1.tv_usec);
    }
    
    /*Write the Shortest Path to a file*/
    results = fopen(argv[6], "w+");
    for (int i = 0; i < 100; i++){
        fprintf(results, "%ld\n", timing[i]);        
        total_time = total_time + timing[i];
    }
    
    printf("Average Time: %ld\n", total_time/100);
    fclose(results);

    //Clean 
    free(sssp_1_h); free(vertex_dim);
    free(vertex_dimT); free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));
    return 0;
}